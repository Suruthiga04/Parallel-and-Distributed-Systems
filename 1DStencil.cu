#include <stdio.h>
#include <hip/hip_runtime.h>
__constant__ int stencil[] = {1, 2, 3,4,5};
//constant stencil array size=radius*2+1

__global__ void stencil_1D(int *input, int *output, int n, int radius) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    int i=0;
    
    for (int offset = -radius; offset <= radius; ++offset) {
        int index = idx + offset;
        if (index >= 0 && index < n) {
            sum += input[index]*stencil[i];
        }
        i++;
    }

    if (idx < n) {
        output[idx] = sum;
    }
}

int main() {
    const int n = 7;
    const int radius = 2;
    int h_input[n] = {5, 2, 1, 9, 2, 3, 6};
    int h_output[n];

    int *d_input, *d_output;

    
    hipMalloc((void**)&d_input, n * sizeof(int));
    hipMalloc((void**)&d_output, n * sizeof(int));

    
    hipMemcpy(d_input, h_input, n * sizeof(int), hipMemcpyHostToDevice);

    
    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;

    
    stencil_1D<<<gridSize, blockSize>>>(d_input, d_output, n, radius);

    hipMemcpy(h_output, d_output, n * sizeof(int), hipMemcpyDeviceToHost);

    
    printf("Input:  ");
    for (int i = 0; i < n; ++i) {
        printf("%d ", h_input[i]);
    }
    printf("\nOutput: ");
    for (int i = 0; i < n; ++i) {
        printf("%d ", h_output[i]);
    }
    printf("\n");

   
    hipFree(d_input);
    hipFree(d_output);

    return 0;
}
