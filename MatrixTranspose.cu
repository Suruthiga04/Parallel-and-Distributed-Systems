#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#define row 4
#define col 4

global void transpose(int *a)
{
int x=blockIdx.x;
int y=blockIdx.y;

shared  int p[col*row];
p[y*row+x]=a[col*x+y];
__syncthreads();
a[y*row+x]=p[y*row+x];
}
int main()
{
int a[row][col]={{1,2,3,4},{1,2,3,4},{1,2,3,4},{1,2,3,4}};
int res[col][row];
int *da;
hipEvent_t start,stop;
hipEventCreate(&start);
hipEventCreate (&stop);

float elapsedTime;
hipMalloc((void **)&da,row*col*sizeof(int));
hipMemcpy(da,&a,row*col*sizeof(int),hipMemcpyHostToDevice);

dim3 grid(row,col);
printf("Before Transpose:\n");
for(int i=0;i<row;i++)
{
for(int j=0;j<col;j++)
{
printf("%d",a[i][j]);
}
printf("\n");
}
hipEventRecord(start,0);
transpose<<<grid,1>>>(da);
hipEventRecord(stop,0);
hipEventElapsedTime(&elapsedTime,start,stop);
printf("\nTime taken for shared memory transpose:%f\n",elapsedTime);
hipMemcpy(&res,da,row*col*sizeof(int),hipMemcpyDeviceToHost);
printf("\nAfter transpose:\n");
for(int i=0;i<row;i++)
{
for(int j=0;j<col;j++)
{
printf("%d",res[i][j]);
}
printf("\n");
}
hipFree(da);
}
