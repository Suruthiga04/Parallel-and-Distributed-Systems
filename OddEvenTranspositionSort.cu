#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#define n 10
_global_ void oddEvenSortKernel(int *a, int phase) {
unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
unsigned int index1 = 2 * i;
unsigned int index2 = index1 + 1;
if (phase % 2 == 0) { // Odd phase
if (index2 < n && index1 < n) {
if (a[index1] > a[index2]) {
int temp = a[index1];
a[index1] = a[index2];
a[index2] = temp;
}
}
} else { // Even phase
if (index1 + 1 < n && index1 + 2 < n) {
if (a[index1 + 1] > a[index1 + 2]) {
int temp = a[index1 + 1];
a[index1 + 1] = a[index1 + 2];
a[index1 + 2] = temp;
}
}
}
}
int main() {
int a[n] = {6, 9, 5, 7, 4, 2, 1, 8, 0, 3};
int *d_a;
// Allocate GPU memory
hipMalloc((void**)&d_a, n * sizeof(int));
hipMemcpy(d_a, a, n * sizeof(int), hipMemcpyHostToDevice);
// Define number of threads and blocks
int numThreads = n / 2;
int numBlocks = 1;
// Perform Odd-Even Transposition Sort
for (int phase = 0; phase < n; ++phase) {
oddEvenSortKernel<<<numBlocks, numThreads>>>(d_a, phase);
hipDeviceSynchronize();
}
// Copy the sorted array back to host
hipMemcpy(a, d_a, n * sizeof(int), hipMemcpyDeviceToHost);
hipFree(d_a);
// Print the sorted array
printf("Sorted Order: ");
for (int i = 0; i < n; i++)
printf("%d ", a[i]);
printf("\n");
return 0;
}
