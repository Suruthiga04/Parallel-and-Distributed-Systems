#include<stdio.h>
#include<hip/hip_runtime.h>
__device__ int temp[9][9];
__global__ void traverse(int *parent,int *child,int *sibling,int *edge0
,int *edge1,int *succ0,int *succ1,int *position,int *preorder) {
int i=threadIdx.x;
if(parent[edge0[i]]==edge1[i]){
 if(sibling[edge0[i]]!=-1){
 succ0[i]=edge1[i];
 succ1[i]=sibling[edge0[i]];
 }
 else if(parent[edge1[i]]!=-1){
 succ0[i]=edge1[i];
 succ1[i]=parent[edge1[i]];
 }
 else{
 succ0[i]=edge0[i];
 succ1[i]=edge1[i];
 preorder[edge1[i]]=1;
}
}
else{
if(child[edge1[i]]!=-1){
succ0[i]=edge1[i];
 succ1[i]=child[edge1[i]];
 }
 else{
 succ0[i]=edge1[i];
 succ1[i]=edge0[i];
 }
}
if(parent[edge0[i]]==edge1[i]){
 position[i]=0;
}
else{
 position[i]=1;
}
int x;
for(int k=0;k<4;k++){
 x=temp[succ0[i]][succ1[i]];
 position[i]=position[i]+position[x];
 succ0[i]=succ0[x];
 succ1[i]=succ1[x];
}
if(edge0[i]==parent[edge1[i]]){
 preorder[edge1[i]]=9+1-position[i];
}
}
__global__ void initialize(int *edge0,int *edge1){
 for(int i=0;i<16;i++){
 temp[edge0[i]][edge1[i]]=i;
 }
}
int main()
{
char vertices[9]={'a','b','c','d','e','f','g','h','i'};
int parent[9]={-1,0,0,1,1,2,3,3,4};
int child[9]={1,3,5,6,8,-1,-1,-1,-1};
int sibling[9]={-1,2,-1,4,-1,-1,7,-1,-1};
int edge0[16]={0,1,1,3,3,6,3,7,1,4,0,2,4,8,2,5};
int edge1[16]={1,0,3,1,6,3,7,3,4,1,2,0,8,4,5,2};
int succ0[16]; int succ1[16]; int position[16]; int preorder[9];
int *dparent,*dchild,*dsibling,*dedge0,*dedge1,*dsucc0,*dsucc1;
int *dposition,*dpreorder;
hipMalloc((void**)&dparent,9*sizeof(int));
hipMalloc((void**)&dchild,9*sizeof(int));
hipMalloc((void**)&dsibling,9*sizeof(int));
hipMalloc((void**)&dedge0,16*sizeof(int));
hipMalloc((void**)&dedge1,16*sizeof(int));
hipMalloc((void**)&dsucc0,16*sizeof(int));
hipMalloc((void**)&dsucc1,16*sizeof(int));
hipMalloc((void**)&dposition,16*sizeof(int));
hipMalloc((void**)&dpreorder,9*sizeof(int));
hipMemcpy(dparent,&parent,9*sizeof(int),hipMemcpyHostToDevice);
hipMemcpy(dchild,&child,9*sizeof(int),hipMemcpyHostToDevice);
hipMemcpy(dsibling,&sibling,9*sizeof(int),hipMemcpyHostToDevice);
hipMemcpy(dedge0,&edge0,16*sizeof(int),hipMemcpyHostToDevice);
hipMemcpy(dedge1,&edge1,16*sizeof(int),hipMemcpyHostToDevice);
hipMemcpy(dsucc0,&succ0,16*sizeof(int),hipMemcpyHostToDevice);
hipMemcpy(dsucc1,&succ1,16*sizeof(int),hipMemcpyHostToDevice);
hipMemcpy(dposition,&position,16*sizeof(int),hipMemcpyHostToDevice);
hipMemcpy(dpreorder,&preorder,9*sizeof(int),hipMemcpyHostToDevice);
initialize<<<1,1>>>(dedge0,dedge1);
traverse<<<1,16>>>(dparent,dchild,dsibling,dedge0,dedge1,dsucc0,dsucc1,
dposition,dpreorder);
hipMemcpy(&succ0,dsucc0,16*sizeof(int),hipMemcpyDeviceToHost);
hipMemcpy(&succ1,dsucc1,16*sizeof(int),hipMemcpyDeviceToHost);
hipMemcpy(&preorder,dpreorder,9*sizeof(int),hipMemcpyDeviceToHost);
printf("Preorder Traversal numbering to the vertices: \n");
for(int i=0;i<9;i++){
 printf("%c -> %d\n",vertices[i],preorder[i]);
}
hipFree(dparent);
hipFree(dchild);
hipFree(dsibling);
hipFree(dedge0);
hipFree(dedge1);
hipFree(dsucc0);
hipFree(dsucc1);
hipFree(dposition);
hipFree(dpreorder);
return 0;
}
