
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
__global__ void mult(int *a,  int *b, int *c, int n){
int tid = threadIdx.x;
if(tid < n){
c[tid] = a[tid]*b[tid];
}
}
__global__ void add(int *c, int n){
int tid = threadIdx.x;
for(int s = 1;s < n;s=s*2){
if(tid % (2*s) == 0 && (tid+s)<n){
c[tid] = c[tid] + c[tid + s];
}
__syncthreads();
}
}
int main(){
int a[4] = {1,2,3,4};
int b[4] = {1,2,3,4};
int c[4]={0,0,0,0};

int n = 4;
int *da, *db, *dc;

hipMalloc(&da, 4*sizeof(int));
hipMalloc(&db, 4*sizeof(int));
hipMalloc(&dc, 4*sizeof(int));

hipMemcpy(da, a, 4*sizeof(int), hipMemcpyHostToDevice);
hipMemcpy(db, b, 4*sizeof(int), hipMemcpyHostToDevice);
mult<<<1,4>>>(da , db , dc, n);
add<<<1,4>>>(dc,n);
hipMemcpy(c, dc, 4*sizeof(int),  hipMemcpyDeviceToHost);
printf("%d\n",c[0]);

hipFree(da);
hipFree(db);
hipFree(dc);
return 0;
}
